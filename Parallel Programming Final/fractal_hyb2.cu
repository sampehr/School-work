#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <malloc.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 256

#define xMin 0.74395
#define xMax 0.74973
#define yMin 0.11321
#define yMax 0.11899


static void CudaTest(char *msg)
{
  hipError_t e;

  hipDeviceSynchronize();
  if (hipSuccess != (e = hipGetLastError())) {
    fprintf(stderr, "%s: %d\n", msg, e);
    fprintf(stderr, "%s\n", hipGetErrorString(e));
    exit(-1);
  }
}

#define mallocOnGPU(addr, size) if (hipSuccess != hipMalloc((void **)&addr, size)) fprintf(stderr, "could not allocate GPU memory\n");  CudaTest("couldn't allocate GPU memory");
#define copyFromGPU(to, from, size) if (hipSuccess != hipMemcpy(to, from, size, hipMemcpyDeviceToHost)) fprintf(stderr, "copying of data from device failed\n");  CudaTest("data copy from device failed");

static __global__ void FractalKernel(int width, int from, int to, int maxdepth, double dx, double dy, unsigned char cnt[])
{  
  if((((threadIdx.x + blockIdx.x * blockDim.x) / width) + from) < to){	
	double cx, cy, x, y, x2, y2;
	int depth;
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
    int row = (index / width) + from;
	int col = index % width;
	
    cy = yMin + row * dy;
    cx = xMin + col * dx;
    x = -cx;
    y = -cy;
    depth = maxdepth;
    do {
      x2 = x * x;
      y2 = y * y;
      y = 2 * x * y - cy;
      x = x2 - y2 - cx;
      depth--;
    } while ((depth > 0) && ((x2 + y2) <= 5.0));
    cnt[(row - from) * width + col] = depth & 255;
  }
}

extern "C" unsigned char *GPU_Init(int size)
{
  // allocate array on GPU and return pointer to it
  unsigned char *cnt_device;
  mallocOnGPU(cnt_device, size);
  
  return cnt_device;
}

extern "C" void GPU_Exec(int width, int from, int to, int maxdepth, double dx, double dy, unsigned char *cnt_d)
{
    int blocks = ((to - from) * width + THREADSPERBLOCK - 1) / THREADSPERBLOCK;
    FractalKernel<<<blocks, THREADSPERBLOCK>>>(width, from, to, maxdepth, dx, dy, cnt_d);
}

extern "C" void GPU_Fini(unsigned char *cnt, unsigned char *cnt_d, int size)
{
  // copy the pixel data to the CPU and deallocate the GPU array
    copyFromGPU(cnt, cnt_d, size); //fourth argument for hipMemcpy present above in #define copyFromGPU
    hipFree(cnt_d);
}
